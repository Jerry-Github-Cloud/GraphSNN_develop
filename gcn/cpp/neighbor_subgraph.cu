#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <ctime>
#include <cmath>
#include <algorithm>
#include <unordered_set>
#include <unordered_map>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;
const int MAX_DEGREE = 50;

/*
G.add_edges_from([
    (0, 1), (0, 2), (0, 3), (0, 4),
    (1, 2), (2, 3), (3, 4)
])

vector<pair<int, int> edge_index {{0,1}, {0,2}, {0,3}, {0,4}, {1,2}, {2,3}, {3,4}};
vector<int> edge_index = {0,1, 0,2, 0,3, 0,4, 1,2, 2,3, 3,4};
*/
vector<int> loadEdgeIndexFromFile(const string& filename) {
    vector<int> edge_index;

    ifstream file(filename);
    if (file.is_open()) {
        string line;

        // Read the first line representing source nodes
        getline(file, line);
        istringstream iss_src(line);

        // Read the corresponding destination nodes from the second line
        getline(file, line);
        istringstream iss_dst(line);

        int source, destination;
        char comma;
        while (iss_src >> source >> comma && iss_dst >> destination >> comma) {
            // edge_index.push_back(make_pair(source, destination));
            // edge_index[0].push_back(source);
            // edge_index[1].push_back(destination);
            edge_index.push_back(source);
            edge_index.push_back(destination);
        }
        iss_src >> source;
        iss_dst >> destination;
        // edge_index.push_back(make_pair(source, destination));
        // edge_index[0].push_back(source);
        // edge_index[1].push_back(destination);
        edge_index.push_back(source);
        edge_index.push_back(destination);
        file.close();
    } else {
        cerr << "Failed to open file: " << filename << endl;
    }

    return edge_index;
}


__global__
void one_hop_subgraph_kernel(
    int num_hops, int* d_edge_index, int num_nodes, int num_edges, int* d_neighbor_subgraph_nodes, int* d_neighbor_subgraph_node_count) {
    
    int node_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (node_idx < num_nodes) {
        // d_neighbor_subgraph_nodes[0] = node_idx;
        int idx = 0;
        d_neighbor_subgraph_nodes[node_idx * MAX_DEGREE + idx] = node_idx;
        d_neighbor_subgraph_node_count[node_idx] ++;
        idx ++;

        for (int i = 0; i < num_edges; i ++) {
            if (node_idx == d_edge_index[2 * i]) {
                d_neighbor_subgraph_nodes[node_idx * MAX_DEGREE + idx] = d_edge_index[2 * i + 1];
                d_neighbor_subgraph_node_count[node_idx] ++;
                idx ++;
            } 
            // else if (node_idx == d_edge_index[2 * i + 1]) {
            //     d_neighbor_subgraph_nodes[node_idx * MAX_DEGREE + idx] = d_edge_index[2 * i];
            //     d_neighbor_subgraph_node_count[node_idx] ++;
            //     idx ++;
            // }
        }
    }
}


vector<vector<int>> get_all_neighbor_subgraph(const int num_nodes, const int num_edges, const vector<int>& edge_index, int num_hops) {
    // cout << "num_nodes = " << num_nodes << endl;
    // cout << "num_edges = " << num_edges << endl;

    int* d_edge_index;
    int* d_neighbor_subgraph_nodes;
    int* d_neighbor_subgraph_node_count;

    hipMalloc((void**)&d_edge_index, sizeof(int) * num_edges * 2);
    hipMalloc((void**)&d_neighbor_subgraph_nodes, sizeof(int) * num_nodes * MAX_DEGREE);
    hipMalloc((void**)&d_neighbor_subgraph_node_count, sizeof(int) * num_nodes);
    hipMemcpy(d_edge_index, edge_index.data(), sizeof(int) * num_edges * 2, hipMemcpyHostToDevice);

    int blockSize = 1024;
    int gridSize = (num_nodes + blockSize - 1) / blockSize;

    one_hop_subgraph_kernel<<<gridSize, blockSize>>>(num_hops, d_edge_index, num_nodes, num_edges, d_neighbor_subgraph_nodes, d_neighbor_subgraph_node_count);

    
    vector<int> h_neighbor_subgraph_node_count(num_nodes, 0);
    hipMemcpy(h_neighbor_subgraph_node_count.data(), d_neighbor_subgraph_node_count, sizeof(int) * num_nodes, hipMemcpyDeviceToHost);
    
    vector<int> h_neighbor_subgraph_nodes(num_nodes * MAX_DEGREE);
    hipMemcpy(h_neighbor_subgraph_nodes.data(), d_neighbor_subgraph_nodes, sizeof(int) * num_nodes * MAX_DEGREE, hipMemcpyDeviceToHost);

    
    hipFree(d_edge_index);
    hipFree(d_neighbor_subgraph_nodes);
    hipFree(d_neighbor_subgraph_node_count);

    vector<vector<int>> all_neighbor_subgraph(num_nodes);
    for (int i = 0; i < h_neighbor_subgraph_nodes.size(); i ++) {
        int central_node_idx = i / MAX_DEGREE;
        int neighbor_node_idx = i % MAX_DEGREE;
        if (neighbor_node_idx < h_neighbor_subgraph_node_count[central_node_idx]) {
            all_neighbor_subgraph[central_node_idx].push_back(h_neighbor_subgraph_nodes[i]);
        }
    }

    return all_neighbor_subgraph;
}


vector<int> intersect(vector<int>& array1, vector<int>& array2) {
    vector<int> result;
    sort(array1.begin(), array1.end());
    sort(array2.begin(), array2.end());
    set_intersection(array1.begin(), array1.end(), array2.begin(), array2.end(), back_inserter(result));
    return result;
}


// pair<int, int> subgraph(const vector<int>& nodes, const vector<pair<int, int>>& edge_index) {
//     unordered_set<int> node_set(nodes.begin(), nodes.end());

//     int num_nodes = 0;
//     int num_edges = 0;

//     for (const auto& [u, v] : edge_index) {
//         if (node_set.count(u) > 0 and node_set.count(v) > 0) {
//             num_edges++;
//         }
//     }
//     num_nodes = node_set.size();
//     return make_pair(num_nodes, num_edges / 2);
// }


pair<int, int> subgraph(const vector<int>& nodes, const vector<int>& edge_index) {
    unordered_set<int> node_set(nodes.begin(), nodes.end());

    int num_nodes = 0;
    int num_edges = 0;

    for (int i = 0; i < num_edges; i ++) {
        int u = edge_index[2 * i], v = edge_index[2 * i + 1];
        if (node_set.count(u) > 0 and node_set.count(v) > 0) {
            num_edges++;
        }
    }
    num_nodes = node_set.size();
    return make_pair(num_nodes, num_edges / 2);
}


pair<int, int> get_overlap_subgraph(vector<int> u_subset, vector<int> v_subset, const vector<int>& edge_index) {
    vector<int> overlapSubset = intersect(u_subset, v_subset);
    return subgraph(overlapSubset, edge_index);
}


// vector<vector<double>> get_sc_matrix(const int num_nodes, const vector<pair<int, int>>& edge_index, int lambda_) {
//     vector<vector<int>> all_neighbor_subgraph = get_all_neighbor_subgraph(num_nodes, edge_index, 1);
//     vector<vector<double>> structural_coeff(num_nodes, vector<double>(num_nodes, 0.));
//     for (const auto [u, v] : edge_index) {
//         const auto [s_num_node, s_num_edge] = get_overlap_subgraph(all_neighbor_subgraph[u], all_neighbor_subgraph[v], edge_index);
//         // structural_coeff[u][v] = s_num_edge * pow(s_num_node, lambda_) / (s_num_node * (s_num_node - 1));
//         // printf("(%d, %d)\ts_num_node=%d\ts_num_edge=%d\n", u, v, s_num_node, s_num_edge);
//         if (s_num_node <= 1) {
//             continue;
//         }
//         double sc = (double)s_num_edge * s_num_node / (s_num_node * (s_num_node - 1));
//         structural_coeff[u][v] = sc;
//     }
//     return structural_coeff;
// }

vector<vector<double>> get_sc_matrix(const int num_nodes, const int num_edges, const vector<int>& edge_index, int lambda_) {
    vector<vector<int>> all_neighbor_subgraph = get_all_neighbor_subgraph(num_nodes, num_edges, edge_index, 1);
    vector<vector<double>> structural_coeff(num_nodes, vector<double>(num_nodes, 0.));
    for (int i = 0; i < num_edges; i ++) {
        int u = edge_index[2 * i], v = edge_index[2 * i + 1];
        const auto [s_num_node, s_num_edge] = get_overlap_subgraph(all_neighbor_subgraph[u], all_neighbor_subgraph[v], edge_index);
        if (s_num_node <= 1) {
            continue;
        }
        double sc = (double)s_num_edge * s_num_node / (s_num_node * (s_num_node - 1));
        structural_coeff[u][v] = sc;
    }
    return structural_coeff;
}


void print_iterable(vector<int> data) {
    for (int x : data) {
        cout << x << " ";
    }
    cout << endl;
    // for (int i = 0; i < data.size(); i ++) {
    //     cout << i << "\t" << data[i] << endl;
    // }
}


int main() {
    // [Demo] load cora edge index
    int num_nodes = 2708;
    string filename = "../out/cora_edge_index.csv";
    vector<int> edge_index = loadEdgeIndexFromFile(filename);
    int num_edges = edge_index.size() / 2;

    // [Test] G1
    // string filename = "../out/G1_edge_index.csv";
    // EdgeIndex edgeIndex = loadEdgeIndexFromFile(filename);
    // for (const auto [u, v] : edgeIndex) {
    //     printf("(%d, %d)\n", u, v);
    // }

    // int num_nodes = 0;
    // for (const auto& edge : edgeIndex) {
    //     num_nodes = max(num_nodes, max(edge.first, edge.second) + 1);
    // }
    // vector<vector<int>> subgraph_list = get_all_neighbor_subgraph(num_nodes, edgeIndex, 1);
    // for (int u = 0; u < num_nodes; u ++) {
    //     print_iterable(subgraph_list[u]);
    // }

    // [Demo] get_all_neighbor_subgraph  
    clock_t start = clock();
    vector<vector<int>> all_neighbor_subgraph = get_all_neighbor_subgraph(num_nodes, num_edges, edge_index, 1);
    clock_t end = clock();
    for (int u = 0; u < num_nodes; u ++) {
        print_iterable(all_neighbor_subgraph[u]);
    }
    
    // double duration = double(end - start) / CLOCKS_PER_SEC;
    // cout << "Execution time: " << duration << " seconds" << endl;

    // auto u_subset = k_hop_subgraph({2706}, 1, edgeIndex);
    // auto v_subset = k_hop_subgraph({2707}, 1, edgeIndex);
    // auto overlap_subset = intersect(u_subset, v_subset);
    // print_iterable(u_subset);
    // print_iterable(v_subset);

    // // [Demo] get_sc_matrix
    // clock_t start = clock();
    // vector<vector<double>> structural_coeff = get_sc_matrix(num_nodes, num_edges, edge_index, 1);
    // clock_t end = clock();
    // double duration = double(end - start) / CLOCKS_PER_SEC;
    // cout << "Execution time: " << duration << " seconds" << endl;

    // int m = structural_coeff.size(), n = structural_coeff[0].size();
    // for (int r = 0; r < m; r ++) {
    //     // cout << "row " << r << ": "; 
    //     for (int c = 0; c < n; c ++) {
    //         if (c == 0) {
    //             printf("%.2f", structural_coeff[r][c]);
    //         } else {
    //             printf(",%.2f", structural_coeff[r][c]);
    //         }
    //         // cout << structural_coeff[r][c] << " ";
    //     }
    //     cout << endl;
    // }
}
